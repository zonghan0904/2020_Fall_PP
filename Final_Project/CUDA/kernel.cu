#include "hip/hip_runtime.h"
# include <hip/hip_runtime.h>
# include <stdio.h>
# include <stdlib.h>
# include <assert.h>
extern "C"{
# include "kernel.h"
}

#define MAX_BRIGHTNESS 255

__global__ void calculate_G(float *G, float *G_x, float *G_y, int width, int height){
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = thisY * width + thisX;

    if (thisX != 0 && thisX != width-1 && thisY != 0 && thisY != height-1){
	G[idx] = (float)hypot(G_x[idx], G_y[idx]);
    }
    else{
	G[idx] = 1;
    }
}

__global__ void non_maximum_sup(float *nms, float *G, float *G_x, float *G_y, int width, int height){
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = thisY * width + thisX;

    if (thisX != 0 && thisX != width-1 && thisY != 0 && thisY != height-1){
       	const int nn = idx - width;
       	const int ss = idx + width;
       	const int ww = idx + 1;
       	const int ee = idx - 1;
       	const int nw = nn + 1;
       	const int ne = nn - 1;
       	const int sw = ss + 1;
       	const int se = ss - 1;
       	const float dir = (float) (fmod(atan2(G_y[idx], G_x[idx]) + M_PI, M_PI) / M_PI) * 8;

        if (((dir <= 1 || dir > 7) && G[idx] > G[ee] && G[idx] > G[ww]) || // 0 deg
            ((dir > 1 && dir <= 3) && G[idx] > G[nw] && G[idx] > G[se]) || // 45 deg
            ((dir > 3 && dir <= 5) && G[idx] > G[nn] && G[idx] > G[ss]) || // 90 deg
            ((dir > 5 && dir <= 7) && G[idx] > G[ne] && G[idx] > G[sw]))   // 135 deg
	    nms[idx] = G[idx];
	else
	    nms[idx] = 0;
    }
    else{
	nms[idx] = 1;
    }
}

__global__ void convolution(float *in, float *out, float *kernel, int nx, int ny, int kn, bool normalize){
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = thisY * nx + thisX;

    const int khalf = kn / 2;
    float min = 0.5;
    float max = 254.5;
    float pixel = 0.0;
    size_t c = 0;
    int i, j;

    assert(kn % 2 == 1);
    assert(nx > kn && ny > kn);

    if (thisX >= khalf && thisX < nx-khalf && thisY >= khalf && thisY < ny-khalf){
	pixel = c = 0;

        for (j = -khalf; j <= khalf; j++)
          for (i = -khalf; i <= khalf; i++)
            pixel += in[(thisY - j) * nx + thisX - i] * kernel[c++];

        if (normalize == true)
          pixel = MAX_BRIGHTNESS * (pixel - min) / (max - min);

        out[idx] = (float) pixel;
    }
    else{
	out[idx] = 0;
    }
}

__global__ void threshold(float *nms, float *thre, int width, int t2){
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = thisY * width + thisX;

    if (nms[idx] >= t2){
	thre[idx] = MAX_BRIGHTNESS;
    }
    else{
	thre[idx] = 0;
    }
}

__global__ void hysteresis(float *nms, float *thre, float *hyster, int width, int height, int t1, int t2){
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = thisY * width + thisX;

    const int nn = idx - width;
    const int ss = idx + width;
    const int ww = idx + 1;
    const int ee = idx - 1;
    const int nw = nn + 1;
    const int ne = nn - 1;
    const int sw = ss + 1;
    const int se = ss - 1;

    hyster[idx] = thre[idx];

    if (thisX != 0 && thisX != width-1 && thisY != 0 && thisY != height-1){
	if (t1 < nms[idx] && nms[idx] < t2){
	    if (thre[ee] != 0 || thre[ww] != 0 ||
		thre[nn] != 0 || thre[ss] != 0 ||
		thre[ne] != 0 || thre[nw] != 0 ||
		thre[se] != 0 || thre[sw] != 0){
		hyster[idx] = MAX_BRIGHTNESS;
	    }
	}
    }
}

/*
 * gaussianFilter: http://www.songho.ca/dsp/cannyedge/cannyedge.html
 * Determine the size of kernel (odd #)
 * 0.0 <= sigma < 0.5 : 3
 * 0.5 <= sigma < 1.0 : 5
 * 1.0 <= sigma < 1.5 : 7
 * 1.5 <= sigma < 2.0 : 9
 * 2.0 <= sigma < 2.5 : 11
 * 2.5 <= sigma < 3.0 : 13 ...
 * kernel size = 2 * int(2 * sigma) + 3;
 */
void gaussian_kernel(float *kernel,
		     const int n,
		     const float mean,
		     const float sigma)
{
  int i, j;
  size_t c = 0;

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++)
      kernel[c++] = exp(-0.5 * (pow((i - mean) / sigma, 2.0) + pow((j - mean) / sigma, 2.0))) / (2 * M_PI * sigma * sigma);
  }
}

/*
 * Links:
 * http://en.wikipedia.org/wiki/Canny_edge_detector
 * http://www.tomgibara.com/computer-vision/CannyEdgeDetector.java
 * http://fourier.eng.hmc.edu/e161/lectures/canny/node1.html
 * http://www.songho.ca/dsp/cannyedge/cannyedge.html
 *
 * Note: T1 and T2 are lower and upper thresholds.
 */
extern "C"
float * canny_edge_detection(const float    *in,
	                     const int      width,
		             const int      height,
			     const int      t1,
			     const int      t2,
			     const float    sigma)
{
  float *retval;
  int dataSize = width * height * sizeof(float);
  const int n = 2 * (int) (2 * sigma) + 3;
  const float mean = (float) floor(n / 2.0);

  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);

  const float Gx[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
  const float Gy[] = {1, 2, 1, 0, 0, 0, -1, -2, -1};

  /* allocate host memory */
  float *kernel = (float*)malloc(n * n * sizeof(float));

  /* allocate device memory */
  float *d_pixels;
  hipMalloc(&d_pixels, dataSize);
  float *d_out;
  hipMalloc(&d_out, dataSize);
  float *d_Gx;
  hipMalloc(&d_Gx, 9 * sizeof(float));
  float *d_Gy;
  hipMalloc(&d_Gy, 9 * sizeof(float));
  float *d_after_Gx;
  hipMalloc(&d_after_Gx, dataSize);
  float *d_after_Gy;
  hipMalloc(&d_after_Gy, dataSize);
  float *d_G;
  hipMalloc(&d_G, dataSize);
  float *d_nms;
  hipMalloc(&d_nms, dataSize);
  float *d_kernel;
  hipMalloc(&d_kernel, n * n * sizeof(float));
  float *d_thre;
  hipMalloc(&d_thre, dataSize);
  float *d_hyster;
  hipMalloc(&d_hyster, dataSize);

  /* copy input data from host to device */
  hipMemcpy(d_pixels, in, dataSize, hipMemcpyHostToDevice);
  hipMemcpy(d_Gx, Gx, 9 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Gy, Gy, 9 * sizeof(float), hipMemcpyHostToDevice);

  /* Gaussian Filter */
  gaussian_kernel(kernel, n, mean, sigma);
  hipMemcpy(d_kernel, kernel, n * n * sizeof(float), hipMemcpyHostToDevice);
  convolution<<<numBlocks, threadsPerBlock>>>(d_pixels, d_out, d_kernel, width, height, n, true);

  /* Sobel Filter */
  convolution<<<numBlocks, threadsPerBlock>>>(d_out, d_after_Gx, d_Gx, width, height, 3, false);
  convolution<<<numBlocks, threadsPerBlock>>>(d_out, d_after_Gy, d_Gy, width, height, 3, false);
  calculate_G<<<numBlocks, threadsPerBlock>>>(d_G, d_after_Gx, d_after_Gy, width, height);

  /* Non-maximum suppression */
  non_maximum_sup<<<numBlocks, threadsPerBlock>>>(d_nms, d_G, d_after_Gx, d_after_Gy, width, height);

  /* threshold */
  threshold<<<numBlocks, threadsPerBlock>>>(d_nms, d_thre, width, t2);

  /* hystersis */
  hysteresis<<<numBlocks, threadsPerBlock>>>(d_nms, d_thre, d_hyster, width, height, t1, t2);

  /* copy output data from device to host */
  retval = (float*)malloc(dataSize);
  hipMemcpy(retval, d_hyster, dataSize, hipMemcpyDeviceToHost);

  /* deallocate both CPU's and GPU's memory */
  hipFree(d_pixels);
  hipFree(d_out);
  hipFree(d_Gx);
  hipFree(d_Gy);
  hipFree(d_after_Gx);
  hipFree(d_after_Gy);
  hipFree(d_G);
  hipFree(d_nms);
  hipFree(d_kernel);
  hipFree(d_thre);
  hipFree(d_hyster);
  free(kernel);

  return retval;
}
