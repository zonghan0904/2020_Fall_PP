#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(int* d_data, float stepX, float stepY, float lowerX, float lowerY, int count, int pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    float c_x = lowerX + thisX * stepX;
    float c_y = lowerY + thisY * stepY;
    float z_x = c_x;
    float z_y = c_y;

    int iter;
    for (iter = 0; iter < count; ++iter){
	if (z_x * z_x + z_y * z_y > 4.f) break;

	float new_x = z_x * z_x - z_y * z_y;
	float new_y = 2.f * z_x * z_y;
	z_x = c_x + new_x;
	z_y = c_y + new_y;
    }

    int* row = (int*)((char*)d_data + thisY * pitch);
    row[thisX] = iter;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int N = resX * resY;
    int size = N * sizeof(int);
    size_t pitch = 0;

    int *data;
    hipHostAlloc(&data, size, hipHostMallocMapped);
    int *d_data;
    hipMallocPitch(&d_data, &pitch, resX * sizeof(int), resY);

    dim3 threadsPerBlock(25, 25);
    dim3 numBlocks(resX / threadsPerBlock.x, resY / threadsPerBlock.y);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(d_data, stepX, stepY, lowerX, lowerY, maxIterations, pitch);

    hipMemcpy2D(data, resX * sizeof(int), d_data, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, data, size);
    hipFree(d_data);
    hipHostFree(data);
}
