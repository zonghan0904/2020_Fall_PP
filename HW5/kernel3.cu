#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(int* d_data, float stepX, float stepY, float lowerX, float lowerY, int count, int pitch, int scale) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * scale;
    int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * scale;

    for (int j = 0; j < scale; j++){
	for (int i = 0; i < scale; i++){
	    float c_x = lowerX + (thisX + i) * stepX;
    	    float c_y = lowerY + (thisY + j) * stepY;
    	    float z_x = c_x;
    	    float z_y = c_y;

    	    int iter;
    	    for (iter = 0; iter < count; ++iter){
    	        if (z_x * z_x + z_y * z_y > 4.f) break;

    	        float new_x = z_x * z_x - z_y * z_y;
    	        float new_y = 2.f * z_x * z_y;
    	        z_x = c_x + new_x;
    	        z_y = c_y + new_y;
    	    }

    	    int* row = (int*)((char*)d_data + (thisY + j) * pitch);
    	    row[thisX + i] = iter;
	}
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int N = resX * resY;
    int size = N * sizeof(int);
    size_t pitch = 0;
    int scale = 4;

    int *data;
    hipHostAlloc(&data, size, hipHostMallocMapped);
    int *d_data;
    hipMallocPitch(&d_data, &pitch, resX * sizeof(int), resY);

    dim3 threadsPerBlock(25, 25);
    dim3 numBlocks(resX / threadsPerBlock.x / scale, resY / threadsPerBlock.y / scale);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(d_data, stepX, stepY, lowerX, lowerY, maxIterations, pitch, scale);

    hipMemcpy2D(data, resX * sizeof(int), d_data, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, data, size);
    hipFree(d_data);
    hipHostFree(data);
}
